#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstddef>
#include <iostream>
#include <numeric>
#include <map>
#include <sys/time.h>
#include <valarray>

#include <hdf5.h>

#include "range.hpp"
#include "utils.hpp"

#define NUM_ROWS 28
#define NUM_COLS 28
#define NUM_CHANNELS 1 // 1 input
#define NUM_DIGITS 10

static int FLAGS_batch_size = 10000;
static std::string FLAGS_testdata{};
static std::string FLAGS_model{};

// Data and reference data dimensions
static int xdims[] = {FLAGS_batch_size, NUM_ROWS, NUM_COLS, NUM_CHANNELS}; //actual input data dimensions
static int rdims[] = {FLAGS_batch_size, NUM_DIGITS}; //reference labels. Each sample in batch contains a vector of NUM_DIGITS size. 

// Model dimensions
static int conv1dims[] = {5, 5, 1, 32}; //5 x 5 filter. 1 input channel featured, 32 output features
static int conv2dims[] = {5, 5, 32, 64}; //32 input features, 64 output features
static int fc1dims[]   = {1024, 128};
static int fc2dims[]   = {128, 10};

static int loadData(float *x, float *y) {
  // Open the data file
  const auto file_id =
      H5Fopen(FLAGS_testdata.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

  // Open the dataset x and y
  const auto x_id = H5Dopen2(file_id, "/x", H5P_DEFAULT);
  const auto y_id = H5Dopen2(file_id, "/y", H5P_DEFAULT);

  // Get the dataset x dimensions
  const auto xspace = H5Dget_space(x_id);
  const auto xndims = H5Sget_simple_extent_ndims(xspace);
  assert(xndims == 4);

  hsize_t input_dims[xndims];
  H5Sget_simple_extent_dims(xspace, input_dims, NULL);
  if (input_dims[0] != FLAGS_batch_size) {
    std::cout << "data size does not match batch size specified!\n";
    return 1; // return error
  }
  std::cout << "input dimensions = " << input_dims[0] << " x " << input_dims[1]
            << " x " << input_dims[2] << " x " << input_dims[3] << "\n";

  // Read the dataset x and y
  check_success(
      H5Dread(x_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, x));
  check_success(
      H5Dread(y_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, y));

  // Close the dataset x and y
  check_success(H5Dclose(x_id));
  check_success(H5Dclose(y_id));

  // Close the file
  check_success(H5Fclose(file_id));

  // return success
  return 0;
}

static void loadModel(float *conv1, float *conv2, float *fc1, float *fc2) {
  // Open the model file
  const auto file_id = H5Fopen(FLAGS_model.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

  // Open the dataset
  const auto conv1_id = H5Dopen2(file_id, "/conv1", H5P_DEFAULT);
  const auto conv2_id = H5Dopen2(file_id, "/conv2", H5P_DEFAULT);
  const auto fc1_id   = H5Dopen2(file_id, "/fc1", H5P_DEFAULT);
  const auto fc2_id   = H5Dopen2(file_id, "/fc2", H5P_DEFAULT);

  // Read the dataset
  check_success(H5Dread(conv1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                        H5P_DEFAULT, conv1));
  check_success(H5Dread(conv2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                        H5P_DEFAULT, conv2));
  check_success(
      H5Dread(fc1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc1));
  check_success(
      H5Dread(fc2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc2));

  // Close the dataset x and y
  check_success(H5Dclose(conv1_id));
  check_success(H5Dclose(conv2_id));
  check_success(H5Dclose(fc1_id));
  check_success(H5Dclose(fc2_id));

  // Close the file
  check_success(H5Fclose(file_id));
}

// Choose the guess with largest score
static void argmax(const float *X, const int xdims[2], int *Y) {
  for (const auto i : range(0, xdims[0])) {
    auto max_idx = 0;
    auto max     = X[i * xdims[1]];
    for (const auto j : range(0, xdims[1])) {
      const auto elem = X[(i * xdims[1]) + j];
      if (elem > max) {
        max_idx = j;
        max     = elem;
      }
    }
    Y[i] = max_idx;
  }
}

  
//Unrolls the input features of a batch specified by the index. This would output an array that is Input Features X_unrolled in figure 16.14 in the notes.
__global__ void unroll_InputOptimized(int C, int H_out, int W_out, int K, int W, float *X, float *X_unroll, int index) 
{
    int t =  blockIdx.x * 1024 + threadIdx.x; //get the thread index which is used as an index to indentify which input feature, and where in the input feature to begin.
    int W_unroll = H_out * W_out;
    int c, s, h_out, w_out, h_unroll, w_unroll, w_base;
    if (t < C * W_unroll)
    {
      c = t % C; //input feature of thread t
      s = t / C; //the linearized position in the input feature to start the double forloop
      h_out = s / W_out; //height of the starting position in the input feature
      w_out = s % W_out; //width of the starting position in the input feature
      h_unroll = s; //which column of the output feature the thread is working on
      w_base = c * K * K;
      //Each thread unrolls K * K elements in the input array and stores it in X_unrolled format. 
      for (const p : range(0, K))
      {
        for (const q : range(0, K))
        {
          w_unroll = w_base + p * K + q;   
          X_unroll[w_unroll * H_out * W_out + h_unroll] = X[index + (h_out + p) * W * C + (w_out + q) * C + c];
        }
      }


    }
}


//Unrolls the weights into W' shown in figure 16.14 of the notes.
__global__ void unroll_W(int C, int M, int K, float * W, float * W_unroll)
{
    int t = blockIdx.x * 1024 + threadIdx.x; //linearized thread index
    if (t < C * M)
    {
      int m = t % M; //output feature of thread t
      int c = t / M; //the cth weight of the mth output feature of thread t
      int unroll_width = C * K * K;
      //unrolls the cth weight of the mth output feature into organization of W.' Each thread unrolls one weight.
      for (const p : range(0, K)) 
      {
        for (const q : range(0, K))
        {
          W_unroll[unroll_width * m + K * K * c + p * K + q] = W[p * K * C * M + q * C * M + c * M + m];
        }
      }
    }
}

//simple matrix multiplication using shared tiled memory. Same as mp.
#define TILE_WIDTH 16
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {

  __shared__ float subTileM[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileN[TILE_WIDTH][TILE_WIDTH];
  
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  
  float Pvalue = 0.0;
  
  for (int i = 0; i < (numAColumns-1)/TILE_WIDTH + 1; i++) {
    
    if ((Row < numARows) && ((i * TILE_WIDTH + threadIdx.x) < numAColumns))
      subTileM[threadIdx.y][threadIdx.x] = A[Row * numAColumns + (i * TILE_WIDTH + threadIdx.x)];
    else
      subTileM[threadIdx.y][threadIdx.x] = 0.0;
    
    if ((Col < numBColumns) && ((i * TILE_WIDTH + threadIdx.y) < numAColumns))
      subTileN[threadIdx.y][threadIdx.x] = B[(i * TILE_WIDTH + threadIdx.y) * numBColumns + Col];
    else
      subTileN[threadIdx.y][threadIdx.x] = 0.0;
    
    __syncthreads();
    for (int j = 0; j < TILE_WIDTH; j++) {
      Pvalue += subTileM[threadIdx.y][j] * subTileN[j][threadIdx.x];
    }
    __syncthreads();
  }
  
  if (Row < numARows && Col < numBColumns) {
    C[Row * numBColumns + Col] = Pvalue;
  }
}


//simple tiled shared matrix multiplication with the caveat that any solution value under 0 is clamped to 0 before it is stored. 
__global__ void matrixMultiply1(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
 
  __shared__ float subTileM[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileN[TILE_WIDTH][TILE_WIDTH];
  
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  
  float Pvalue = 0.0;
  
  // figure this shit out
  for (int i = 0; i < (numAColumns-1)/TILE_WIDTH + 1; i++) {
    
    if ((Row < numARows) && ((i * TILE_WIDTH + threadIdx.x) < numAColumns))
      subTileM[threadIdx.y][threadIdx.x] = A[Row * numAColumns + (i * TILE_WIDTH + threadIdx.x)];
    else
      subTileM[threadIdx.y][threadIdx.x] = 0.0;
    
    if ((Col < numBColumns) && ((i * TILE_WIDTH + threadIdx.y) < numAColumns))
      subTileN[threadIdx.y][threadIdx.x] = B[(i * TILE_WIDTH + threadIdx.y) * numBColumns + Col];
    else
      subTileN[threadIdx.y][threadIdx.x] = 0.0;
    
    __syncthreads();
    for (int j = 0; j < TILE_WIDTH; j++) {
      Pvalue += subTileM[threadIdx.y][j] * subTileN[j][threadIdx.x];
    }
    __syncthreads();
  }
  
  if (Row < numARows && Col < numBColumns) {
    if (Pvalue < 0)
      C[Row * numCColumns + Col] = 0;
  else C[Row * numCColumns + Col] = Pvalue;
  }
}


//Places the solution of the matrix multiplication, Y, in figure 16.14 in the notes with the correct dimension data order into the final output. 
__global__ void placeIntoY(float *Y_unroll, float *deviceY, int H, int W, int index)
{
    int t =  blockIdx.x * 1024 + threadIdx.x; //linearized thread index.
    if (t < W * H)
    {
      int h = t / W; //height of the Y element to be placed into output, deviceY.
      int w = t % W; //width of the Y element to be placed into output, deviceY.

      //clamping 
      if (Y_unroll[w + h * W] < 0)
      {
        deviceY[index + w * H + h] = 0;
      }
      else deviceY[index + w * H + h] = Y_unroll[w + h * W];
      
     
    }

}

//host code that implements convolution layers. 
void convLayer_forward(int xdims[4], int wdims[4], float* X, float* Y, float* W)
{

    float *deviceX; //holds the input data of all batches
    float *deviceX2;
    float *deviceX3;
    float *deviceX4;

    float *deviceW; //holds the input data of all the weights

    float *deviceY; //holds the final output data
    float *deviceY2;
    float *deviceY3;
    float *deviceY4;

    float *deviceUnrollX; //holds the first operand of the input feature unrolled
    float *deviceUnrollX2;
    float *deviceUnrollX3;
    float *deviceUnrollX4;

    float *deviceUnrollW; //holds the unrolled matrix of all weights

    float *deviceUnrollY; //Holds the unrolled output of the matrix multiplcition
    float *deviceUnrollY2;
    float *deviceUnrollY3;
    float *deviceUnrollY4;

    int H_out = xdims[1] - wdims[0] + 1; //height of output feature
    int W_out = xdims[2] - wdims[1] + 1; //width of output feature

    int M = wdims[3]; //number of output features
    int C = wdims[2]; //number of input features
    int N = xdims[0]; //number of batches

    //hipMalloc((void**) &deviceX, xdims[1] * xdims[2] * xdims[3] * sizeof(float));
    hipMalloc((void**) &deviceX, xdims[0] * xdims[1] * xdims[2] * xdims[3] * sizeof(float));

    hipMalloc((void**) &deviceW, wdims[0] * wdims[1] * wdims[2] *  wdims[3] * sizeof(float));
    hipMalloc((void**) &deviceY, xdims[0] * (xdims[1] - wdims[0] + 1) * (xdims[2] - wdims[1] + 1) *  wdims[3] * sizeof(float));
    //Xcheck = (float *)malloc(xdims[1] * xdims[2] * xdims[3] * sizeof(float));


    hipMalloc((void**) &deviceX2, xdims[1] * xdims[2] * xdims[3] * sizeof(float));
    hipMalloc((void**) &deviceX3, xdims[1] * xdims[2] * xdims[3] * sizeof(float));
    hipMalloc((void**) &deviceX4, xdims[1] * xdims[2] * xdims[3] * sizeof(float));

    hipMalloc((void**) &deviceY2, xdims[0] * (xdims[1] - wdims[0] + 1) * (xdims[2] - wdims[1] + 1) *  wdims[3] * sizeof(float));
    hipMalloc((void**) &deviceY3, xdims[0] * (xdims[1] - wdims[0] + 1) * (xdims[2] - wdims[1] + 1) *  wdims[3] * sizeof(float));
    hipMalloc((void**) &deviceY4, xdims[0] * (xdims[1] - wdims[0] + 1) * (xdims[2] - wdims[1] + 1) *  wdims[3] * sizeof(float));
    

    hipMalloc((void**) &deviceUnrollY, M * H_out * W_out * sizeof(float));
    hipMalloc((void**) &deviceUnrollX, H_out * W_out * (wdims[0] * wdims[1] * C) * sizeof(float));
    hipMalloc((void**) &deviceUnrollW, wdims[0] * wdims[1] * wdims[2] *  wdims[3] * sizeof(float));


    hipMalloc((void**) &deviceUnrollY2, M * H_out * W_out * sizeof(float));
    hipMalloc((void**) &deviceUnrollX2, H_out * W_out * (wdims[0] * wdims[1] * C) * sizeof(float));
    hipMalloc((void**) &deviceUnrollY3, M * H_out * W_out * sizeof(float));
    hipMalloc((void**) &deviceUnrollX3, H_out * W_out * (wdims[0] * wdims[1] * C) * sizeof(float));
    hipMalloc((void**) &deviceUnrollY4, M * H_out * W_out * sizeof(float));
    hipMalloc((void**) &deviceUnrollX4, H_out * W_out * (wdims[0] * wdims[1] * C) * sizeof(float));
    
    int X_height = C * wdims[0] * wdims[1]; //unrolled input feature height
    int X_width = H_out * W_out; //unrolled input feature width

    int W_width = X_height; //unrolled weight matrix width
    int W_height = M; //unrolled weight mtrix height
    

    int Y_height = W_height; //product of matrix height
    int Y_width = X_width; //product of matrix width
    
    //generate block and thread dimensions for device functions

    dim3 DimBlock(1024, 1, 1);

    int num_threadsInput = C * H_out * W_out;
    int num_blocksInput =  ceil((num_threadsInput + 1023) / 1024);
    dim3 DimGridInput(num_blocksInput, 1, 1);

    int num_threadsW = C * M;
    int num_blocksW = ((num_threadsW + 1023) / 1024);
    dim3 DimGridW(num_blocksW, 1, 1);

    dim3 DimBlockMultiply(16, 16, 1);
    int x = (Y_width+ 16 - 1)/16;
    int y = (Y_height + 16 - 1)/16;
    dim3 DimGridMultiply(x, y, 1);

    int num_threadsY = H_out * W_out * M;
    int num_blocksY = ((num_threadsY + 1023) / 1024);
    dim3 DimGridY(num_blocksY, 1, 1);

    hipMemcpy(deviceW, W, wdims[0] * wdims[1] * wdims[2] *  wdims[3] * sizeof(float), hipMemcpyHostToDevice);
   
    //unroll weights
    unroll_W<<<DimGridW, DimBlock>>>(C, M, wdims[0], deviceW, deviceUnrollW);

    
    hipStream_t stream0;
    hipStream_t stream1;
    hipStream_t stream2;
    hipStream_t stream3;

    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    hipMemcpy(deviceX, X, xdims[0] * xdims[1] * xdims[2] * xdims[3] * sizeof(float), hipMemcpyHostToDevice);

    //go through each iteration. Calculates four batches in each iteration
    for (auto i = 0; i < N; i+=4)
    {
        int index = i * xdims[1] * xdims[2] * xdims[3]; //index of the current iteration for input data
        int index1 = (i+1) * xdims[1] * xdims[2] * xdims[3];
        int index2 = (i+2) * xdims[1] * xdims[2] * xdims[3];
        int index3 = (i+3) * xdims[1] * xdims[2] * xdims[3];

        int yindex = i * H_out * W_out * M; //index of the current iteration for output data
        int yindex1 = (i+1) * H_out * W_out * M;
        int yindex2 = (i+2) * H_out * W_out * M;
        int yindex3 = (i+3) * H_out * W_out * M;

        unroll_InputOptimized<<<DimGridInput, DimBlock, 0, stream0>>>(C, H_out, W_out, wdims[0], xdims[2], deviceX, deviceUnrollX, index);
        unroll_InputOptimized<<<DimGridInput, DimBlock, 0, stream1>>>(C, H_out, W_out, wdims[0], xdims[2], deviceX, deviceUnrollX2, index1);  
        unroll_InputOptimized<<<DimGridInput, DimBlock, 0, stream2>>>(C, H_out, W_out, wdims[0], xdims[2], deviceX, deviceUnrollX3, index2);  
        unroll_InputOptimized<<<DimGridInput, DimBlock, 0, stream3>>>(C, H_out, W_out, wdims[0], xdims[2], deviceX, deviceUnrollX4, index3);     
  
        
        matrixMultiply<<<DimGridMultiply, DimBlockMultiply, 0, stream0>>> (deviceUnrollW, deviceUnrollX, deviceUnrollY, W_height, W_width, X_height, 
                                        X_width, Y_height, Y_width);

        matrixMultiply<<<DimGridMultiply, DimBlockMultiply, 0, stream1>>> (deviceUnrollW, deviceUnrollX2, deviceUnrollY2, W_height, W_width, X_height, 
                                        X_width, Y_height, Y_width);     

        matrixMultiply<<<DimGridMultiply, DimBlockMultiply, 0, stream2>>> (deviceUnrollW, deviceUnrollX3, deviceUnrollY3, W_height, W_width, X_height, 
                                        X_width, Y_height, Y_width);  

        matrixMultiply<<<DimGridMultiply, DimBlockMultiply, 0, stream3>>> (deviceUnrollW, deviceUnrollX4, deviceUnrollY4, W_height, W_width, X_height, 
                                        X_width, Y_height, Y_width);  

        placeIntoY<<<DimGridY, DimBlock, 0, stream0>>>(deviceUnrollY, deviceY, M, H_out * W_out, yindex);
        placeIntoY<<<DimGridY, DimBlock, 0, stream1>>>(deviceUnrollY2, deviceY, M, H_out * W_out, yindex1); 
        placeIntoY<<<DimGridY, DimBlock, 0, stream2>>>(deviceUnrollY3, deviceY, M, H_out * W_out, yindex2); 
        placeIntoY<<<DimGridY, DimBlock, 0, stream3>>>(deviceUnrollY4, deviceY, M, H_out * W_out, yindex3); 
   
    } 
    
     hipMemcpy(Y, deviceY, N * H_out * W_out * M * sizeof(float), hipMemcpyDeviceToHost);   
}

//Each thread average a two by two square for each output element. Each thread does one output elements in all features and batches
__global__ void subsample(float *deviceInput, int inputH, int inputW, int outputW, int outputH, float *deviceOutput, int poolsize, int M, 
  int inputsize, int outputsize, int N)
{
  int t =  blockIdx.x * 1024 + threadIdx.x; //thread index
  if (t < (outputH * outputW * M * N))
  {
    int index = t/(outputH * outputW * M); //Specifies which batch this thread is working on
    int m = (t%(outputH * outputW * M)) % M; //get output feature based on thread
    int distance = (t%(outputH * outputW * M)) / M * poolsize; 
    int w = distance % inputW; //width index of the top left corner of the square
    int h = (distance / inputW) * poolsize; //height index of the top left corner of the square
    float sum = 0;
    //go through the entire square and calculate average
    for (const p : range(0, poolsize))
      {
        for (const q : range(0, poolsize))
        {
          sum += deviceInput[(index * inputsize) + (h + p) * inputH * M + (w + q) * M + m]/ float(poolsize * poolsize);
        }
      }

      deviceOutput[(index * outputsize) + (h/poolsize) * outputH * M + (w/poolsize) * M + m] = sum;
  }
}

//host code that calls subsample device code
void subsampling_layer(float *input, float *output, int poolsize, int inputdims[4], int outputdims[4])
{
    float * deviceX; //input feature data data
    float * deviceY; //output feature data

    int N = inputdims[0]; //number of batches
    int H_in = inputdims[1]; //input feature height
    int W_in = inputdims[2]; //input feature width

    int H_out = outputdims[1]; //output feature height
    int W_out = outputdims[2]; //output feature width


    hipMalloc((void**) &deviceX, inputdims[0] * inputdims[1] * inputdims[2] * inputdims[3] * sizeof(float));
    hipMalloc((void**) &deviceY, outputdims[0] * outputdims[1] * outputdims[2] * outputdims[3] * sizeof(float));

    dim3 DimBlockSample(1024, 1, 1);
    int num_threadsInput = outputdims[1] * outputdims[2] * outputdims[3] * N;
    int num_blocksInput =  ceil((num_threadsInput + 1023) / 1024);

    dim3 DimGridSample(num_blocksInput, 1, 1);
    hipMemcpy(deviceX, input, inputdims[0] * inputdims[1] * 
        inputdims[2] * inputdims[3] * sizeof(float), hipMemcpyHostToDevice);
    
    int inputsize = inputdims[1] * inputdims[2] * inputdims[3];
    int outputsize = outputdims[1] * outputdims[2] * outputdims[3];


    //subsamples the entire input data and put in deviceY
    subsample<<<DimGridSample, DimBlockSample>>>(deviceX, H_in, W_in, W_out, H_out, deviceY, poolsize, inputdims[3], inputsize, outputsize, N);


    //copy deviceY into output host memory
    hipMemcpy(output, deviceY, outputdims[0] * outputdims[1] * outputdims[2] * outputdims[3] * sizeof(float), hipMemcpyDeviceToHost);   
}


//Host code to call the memory multiplication
void fully_forward(const float *X, const int xdims[2], float *W,
                          const int wdims[2], float *Y, const int ydims[2], int check) {

  int numARows = xdims[0], numAColumns = xdims[1];
  int numBRows = wdims[0], numBColumns = wdims[1];
  int numCRows = ydims[0], numCColumns = ydims[1];
  
  float *deviceA;
  float *deviceB;
  float *deviceC;

  //@@ Allocate GPU memory here
  hipMalloc((void**) &deviceA, numARows*numAColumns*sizeof(float));
  hipMalloc((void**) &deviceB, numBRows*numBColumns*sizeof(float));
  hipMalloc((void**) &deviceC, numCRows*numCColumns*sizeof(float));

  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, X, numARows*numAColumns*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, W, numBRows*numBColumns*sizeof(float), hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(numCColumns/16.0), ceil(numCRows / 16.0), 1);
  if(numCColumns % 16) {
    DimGrid.x++;
  }
  if(numCRows % 16) {
    DimGrid.y++;
  }
  dim3 DimBlock(16, 16, 1);

  //@@ Launch the GPU Kernel here
  if (check == 1)
    matrixMultiply1<<<DimGrid,DimBlock>>>(deviceA,deviceB,deviceC,numARows,
                               numAColumns, numBRows,
                               numBColumns, numCRows,
                               numCColumns);
  else matrixMultiply<<<DimGrid,DimBlock>>>(deviceA,deviceB,deviceC,numARows,
                               numAColumns, numBRows,
                               numBColumns, numCRows,
                               numCColumns);

  hipDeviceSynchronize();

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(Y, deviceC, numCRows*numCColumns*sizeof(float), hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
}

// Forward operation for the CNN, a combination of conv layer + average pooling
// + relu
void forward_operation(float *x, float *conv1, float *conv2, float *fc1,
                       float *fc2, int *out) {

  // conv layer
  int adims[] = {xdims[0], (xdims[1] - conv1dims[0] + 1),
                       (xdims[2] - conv1dims[1] + 1), conv1dims[3]}; //batch size, output_height, output_width, number of output features
  //auto a = zeros<float>(adims);
  float * a = (float*)malloc(xdims[0] * (xdims[1] - conv1dims[0] + 1) *
                       (xdims[2] - conv1dims[1] + 1) * conv1dims[3]*sizeof(float));
  //conv_forward_valid(x, xdims, conv1, conv1dims, a, adims);
 convLayer_forward(xdims, conv1dims, x, a, conv1);

  // average pooling
  const int pool_size = 2;
  int bdims[]   = {adims[0], adims[1] / pool_size, adims[2] / pool_size,
                       adims[3]};

  //auto b = zeros<float>(bdims);
  float * b = (float*)malloc(adims[0] * adims[1] / pool_size * adims[2] / pool_size *
                       adims[3]*sizeof(float));
  //average_pool(a, adims, pool_size, b, bdims);
  subsampling_layer(a, b, pool_size, adims, bdims);

  // conv layer
  int cdims[] = {bdims[0], (bdims[1] - conv2dims[0] + 1),
                       (bdims[2] - conv2dims[1] + 1), conv2dims[3]};

  //auto c = zeros<float>(cdims);
  float * c = (float*)malloc(bdims[0]* (bdims[1] - conv2dims[0] + 1)*
                       (bdims[2] - conv2dims[1] + 1)* conv2dims[3]*sizeof(float));
  //conv_forward_valid(b, bdims, conv2, conv2dims, c, cdims);

 convLayer_forward(bdims, conv2dims, b, c, conv2);

  // average pooling
  int ddims[] = {cdims[0], cdims[1] / pool_size, cdims[2] / pool_size,
                       cdims[3]};
  //auto d = zeros<float>(ddims);
  float * d = (float*)malloc(cdims[0] * cdims[1] / pool_size * cdims[2] / pool_size *
                       cdims[3]*sizeof(float));

  //average_pool(c, cdims, pool_size, d, ddims);

  subsampling_layer(c, d, pool_size, cdims, ddims);


  // reshape
  const int ddims2[] = {ddims[0], ddims[1] * ddims[2] * ddims[3]};

  // matrix multiplication
  const int edims[] = {ddims[0], fc1dims[1]};
  

  //auto e            = zeros<float>(edims);
  float * e = (float*)malloc(ddims[0] * fc1dims[1]*sizeof(float));

  fully_forward(d, ddims2, fc1, fc1dims, e, edims, 1);

  // matrix multiplication
  const int fdims[] = {edims[0], fc2dims[1]};
  //auto f            = zeros<float>(fdims);
  float * f = (float*)malloc(edims[0]* fc2dims[1]*sizeof(float));

  fully_forward(e, edims, fc2, fc2dims, f, fdims, 0);

  argmax(f, fdims, out);

  delete[] a;
  delete[] b;
  delete[] c;
  delete[] d;
  delete[] e;
  delete[] f;
}

int main(int argc, char **argv) {

  //size_t sz = 1048576 * 4;
 //hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);

  if (argc != 3 && argc != 4) {
    std::cerr << "\n"
              << "This program performs the forward opertion step for "
                 "Convolutional Neural Network(CNN).  "
                 "Sample usage: \n"
              << argv[0]
              << " [../data/test10.hdf5] [../data/model.hdf5] [10]\n";
    return -1;
  }
  FLAGS_testdata = std::string(argv[1]);
  FLAGS_model    = std::string(argv[2]);
  if (argc == 3) {
    const std::map<std::string, int> default_batch_sizes{
        {"../data/test2.hdf5", 2},
        {"../data/test10.hdf5", 10},
        {"../data/test100.hdf5", 100},
        {"../data/testfull.hdf5", 10000}};
    const auto batch_size_in_map = default_batch_sizes.find(FLAGS_testdata);
    if (batch_size_in_map == default_batch_sizes.end()) {
      std::cerr << "\nERROR:: Unrecognized file " << FLAGS_testdata << " batch_size must be specified.\n";
      return -1;
    }
    FLAGS_batch_size = batch_size_in_map->second;
  } else if (argc == 4) {
    FLAGS_batch_size = atoi(argv[3]);
  }
  xdims[0] = FLAGS_batch_size;
  rdims[0] = FLAGS_batch_size;

  // Load data into x and y
  float *x = allocate<float>(xdims); //x contains data for images
  float *y = allocate<float>(rdims); //y contains label vectors of images
  loadData(x, y);

  // Load model
  float *conv1 = allocate<float>(conv1dims);
  float *conv2 = allocate<float>(conv2dims);
  float *fc1   = allocate<float>(fc1dims);
  float *fc2   = allocate<float>(fc2dims);
  loadModel(conv1, conv2, fc1, fc2);

  // Perform forward opertion
  int *out = zeros<int>(FLAGS_batch_size); //contains predicted labels for each sample

  // get start time
  const auto start = now();

  forward_operation(x, conv1, conv2, fc1, fc2, out);

  // get end time
  const auto end = now();

  // get elapsed time in milliseconds
  const auto elapsed =
      std::chrono::duration<double, std::milli>(end - start).count();

  // Get reference
  int *ref = zeros<int>(FLAGS_batch_size); //contains true labels for each sample
  argmax(y, rdims, ref);

  // Calculate correctness
  int num_correct = 0;

  
  for (const auto i : range(0, FLAGS_batch_size)) {
    if (out[i] == ref[i]) {
      num_correct++;
    }
  }
  
  std::cout << "Done with " << FLAGS_batch_size << " queries in "
            << "elapsed = " << elapsed << " milliseconds. Correctness: "
            << static_cast<float>(num_correct) / FLAGS_batch_size << "\n";

  delete[] x;
  delete[] y;
  delete[] conv1;
  delete[] conv2;
  delete[] fc1;
  delete[] fc2;
  delete[] out;
  delete[] ref;

  return 0;
}
